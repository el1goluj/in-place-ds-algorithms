/***************************************************************************
 *cr
 *cr            (C) Copyright 2015 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
/*
  In-Place Data Sliding Algorithms for Many-Core Architectures, presented in ICPP’15

  Copyright (c) 2015 University of Illinois at Urbana-Champaign. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Authors: Juan Gómez-Luna (el1goluj@uco.es, gomezlun@illinois.edu), Li-Wen Chang (lchang20@illinois.edu)
*/

#include "../../DS/ds.h"
#include "kernel.cu"

// Sequential CPU version
void cpu_streamcompaction(T* input, int elements, T value){
  int j = 0;
  for (int i = 0; i < elements; i++){
    if (input[i] != value){
      input[j] = input[i];
      j++;		
    }
  }
}

int main(int argc, char **argv){

  // Syntax verification
  if (argc != 4) {
      printf("Wrong format\n");
      printf("Syntax: %s <Device Input (%% elements) numElements>\n",argv[0]);
      exit(1);
  }
  int device = atoi(argv[1]);
  int input = atoi(argv[2]);
  int numElements = atoi(argv[3]);
  size_t size = numElements * sizeof(T);
  int value = 0; // Value to remove

  // Set device
  hipDeviceProp_t device_properties;
  hipGetDeviceProperties(&device_properties,device);
  hipSetDevice(device);

  printf("DS Stream Compaction on %s\n", device_properties.name);
  printf("Thread block size = %d\n", L_DIM);
  printf("Coarsening factor = %d\n", REGS);
#ifdef FLOAT
  printf("Single precision array: %d elements\n", numElements);
#elif INT
  printf("Integer array: %d elements\n", numElements);
#else
  printf("Double precision array: %d elements\n", numElements);
#endif

  // Event creation
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float time1 = 0;
  float time2 = 0;

  // Allocate the host input vector A
  T *h_A = (T*)malloc(size);

  // Allocate the host output vector
  T *h_B = (T*)malloc(size);

  // Allocate the device input vector A
  T *d_A = NULL;
  hipMalloc((void **)&d_A, size);

#define WARMUP 0
#define REP 1
  for(int iteration = 0; iteration < REP+WARMUP; iteration++){
    // Initialize the host input vector
    srand(2014);
    for(int i = 0; i < numElements; i++)
    	h_A[i] = value;
    int M = (numElements * input)/100;
    int m = M;
    while(m>0){
        int x = (int)(numElements*(((float)rand()/(float)RAND_MAX)));
        if(h_A[x]==value){
    	    h_A[x] = x+2;
            m--;
        }
    }

#if PRINT
    for(int i = 0; i < numElements; ++i){
        printf("%d ",*(h_A+i));
    }
    printf("\n");
#endif

    // Copy the host input vector A in host memory to the device input vector in device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    int ldim = L_DIM;
    // Atomic flags
    unsigned int* d_flags = NULL;
    int num_flags = numElements % (ldim * REGS) == 0 ? numElements / (ldim * REGS) : numElements / (ldim * REGS) + 1;
    unsigned int *flags = (unsigned int *)calloc(sizeof(unsigned int), num_flags + 2);
    flags[0] = 1;
    flags[num_flags + 1] = 0;
    hipMalloc((void **)&d_flags, (num_flags + 2) * sizeof(unsigned int));
    hipMemcpy(d_flags, flags, (num_flags + 2) * sizeof(unsigned int), hipMemcpyHostToDevice);
    free(flags);
    // Number of work-groups/thread blocks
    int num_wg = num_flags;

    // Start timer
    hipEventRecord( start, 0 );

    // Kernel launch
    streamcompaction<<<num_wg, ldim>>>(d_A, d_A, numElements, d_flags, value);

    unsigned int flagM = 0;
    hipMemcpy(&flagM, d_flags + num_flags, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // End timer
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time1, start, stop );
    if(iteration >= WARMUP) time2 += time1;

    if(iteration == REP+WARMUP-1){
      float timer = time2 / REP;
      double bw = (double)((numElements + numElements*((double)input/100)) * sizeof(T)) / (double)(timer * 1000000.0);
      printf("Execution time = %f ms, Throughput = %f GB/s\n", timer, bw);
    }

    // Free flags
    hipFree(d_flags);
  }
  // Copy to host memory
  hipMemcpy(h_B, d_A, size, hipMemcpyDeviceToHost);

  // CPU execution for comparison
  cpu_streamcompaction(h_A, numElements, value);

  // Verify that the result vector is correct
#if PRINT
  for(int i = 0; i < numElements; ++i){
     printf("%d ",*(h_B+i));
  }
  printf("\n");
  for(int i = 0; i < numElements; ++i){
      printf("%d ",*(h_A+i));
  }
  printf("\n");
#endif
  for (int i = 0; i < (numElements * input)/100; ++i){
      if (h_B[i] != h_A[i]){
          fprintf(stderr, "Result verification failed at element %d!\n", i);
          exit(EXIT_FAILURE);
      }
  }
  printf("Test PASSED\n");

  // Free device global memory
  hipFree(d_A);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  // Free host memory
  free(h_A);
  free(h_B);

  return 0;
}

