/***************************************************************************
 *cr
 *cr            (C) Copyright 2015 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
/*
  In-Place Data Sliding Algorithms for Many-Core Architectures, presented in ICPP’15

  Copyright (c) 2015 University of Illinois at Urbana-Champaign. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Authors: Juan Gómez-Luna (el1goluj@uco.es, gomezlun@illinois.edu), Li-Wen Chang (lchang20@illinois.edu)
*/

#include "../../DS/ds.h"

// Sample predicate for partition (only for INT)
struct is_even{
  __host__ __device__
  bool operator()(const T &x){
    return (x % 2) == 0;
  }
};

#include "kernel.cu"

// Sequential CPU version
void cpu_partition(T* input, int elements, struct is_even pred){
  T* aux = (T*)malloc(sizeof(T)*elements);
  int pos1 = 0;
  int pos2 = 0;
  for (int i = 0; i < elements; i++){
    if(pred(input[i])){
	  input[pos1] = input[i];
      pos1++;
    }
    else{
      aux[pos2] = input[i];
      pos2++;
    }
  }
  for (int i = 0; i < pos2; i++){
	input[pos1 + i] = aux[i];
  }
}

int main(int argc, char **argv){

  // Syntax verification
  if (argc != 4) {
      printf("Wrong format\n");
      printf("Syntax: %s <Device Input (%% elements) numElements>\n",argv[0]);
      exit(1);
  }
  int device = atoi(argv[1]);
  int input = atoi(argv[2]);
  int numElements = atoi(argv[3]);
  size_t size = numElements * sizeof(T);

  // Set device
  hipDeviceProp_t device_properties;
  hipGetDeviceProperties(&device_properties,device);
  hipSetDevice(device);

  printf("DS Partition on %s\n", device_properties.name);
  printf("Thread block size = %d\n", L_DIM);
  printf("Coarsening factor = %d\n", REGS);
#ifdef FLOAT
  printf("Single precision array: %d elements\n", numElements);
#elif INT
  printf("Integer array: %d elements\n", numElements);
#else
  printf("Double precision array: %d elements\n", numElements);
#endif

  // Event creation
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float time1 = 0;
  float time2 = 0;

  // Allocate the host input vector A
  T *h_A = (T*)malloc(size);

  // Allocate the host output vector
  T *h_B = (T*)malloc(size);

  // Allocate the device input vector A and auxiliary vector B
  T *d_A = NULL;
  hipMalloc((void **)&d_A, size);
  T *d_B = NULL;
  hipMalloc((void **)&d_B, size);

#define WARMUP 2
#define REP 10
  for(int iteration = 0; iteration < REP+WARMUP; iteration++){
    // Initialize the host input vectors
    srand(2014);
    for(int i = 0; i < numElements; i++)
    	h_A[i] = i % 2 != 0 ? i:i+1;
    int M = (numElements * input)/100;
    int m = M;
    while(m>0){
        int x = (int)(numElements*(((float)rand()/(float)RAND_MAX)));
        if(h_A[x] % 2 != 0){
    	    h_A[x] = x * 2;
            m--;
        }
    }

#if PRINT
    printf("\n");
    for(int i = 0; i < numElements; ++i){
        printf("%d ",*(h_A+i));
    }
    printf("\n");
#endif

    // Copy the host input vector A in host memory to the device input vector in device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    int ldim = L_DIM;
    // Atomic flags
    unsigned int* d_flags1 = NULL;
    unsigned int* d_flags2 = NULL;
    const int num_flags = numElements % (ldim * REGS) == 0 ? numElements / (ldim * REGS) : numElements / (ldim * REGS) + 1;
    unsigned int *flags = (unsigned int *)calloc(sizeof(unsigned int), num_flags + 2);
    flags[0] = 1;
    flags[num_flags + 1] = 0;
    hipMalloc((void **)&d_flags1, (num_flags + 2) * sizeof(unsigned int));
    hipMemcpy(d_flags1, flags, (num_flags + 2) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_flags2, (num_flags + 2) * sizeof(unsigned int));
    hipMemcpy(d_flags2, flags, (num_flags + 2) * sizeof(unsigned int), hipMemcpyHostToDevice);
    free(flags);
    // Number of work-groups/thread blocks
    int num_wg = num_flags;

    // Start timer
    hipEventRecord( start, 0 );

    // Kernel launch
    partition<<<num_wg, ldim>>>(d_A, d_B, d_A, numElements, d_flags1, d_flags2, is_even());

    unsigned int flagM = 0;
    hipMemcpy(&flagM, d_flags1 + num_flags, sizeof(unsigned int), hipMemcpyDeviceToHost);
    num_wg = (numElements - flagM + 1) == 0 ? 1 : (numElements - flagM + 1) % ldim == 0 ? (numElements - flagM + 1) / ldim : (numElements - flagM + 1) / ldim + 1;
    hipDeviceSynchronize();
    if((numElements - flagM + 1) != 0)
      move_part<<<num_wg, ldim>>>(d_A, d_B, flagM - 1, numElements);
      //hipMemcpy(&d_A[flagM - 1], d_B, (numElements - flagM + 1)*sizeof(int), hipMemcpyDeviceToDevice);

    // End timer
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time1, start, stop );
    if(iteration >= WARMUP) time2 += time1;

    if(iteration == REP+WARMUP-1){
      float timer = time2 / REP;
      double bw = (double)((2 * numElements) * sizeof(T)) / (double)(timer * 1000000.0);
      printf("Execution time = %f ms, Throughput = %f GB/s\n", timer, bw);
    }

    // Free flags
    hipFree(d_flags1);
    hipFree(d_flags2);
  }
  // Copy to host memory
  hipMemcpy(h_B, d_A, size, hipMemcpyDeviceToHost);

  // CPU execution for comparison
  cpu_partition(h_A, numElements, is_even());

  // Verify that the result vector is correct
#if PRINT
  for(int i = 0; i < numElements; ++i){
     printf("%d ",*(h_B+i));
  }
  printf("\n");
  for(int i = 0; i < numElements; ++i){
      printf("%d ",*(h_A+i));
  }
  printf("\n");
#endif
  for (int i = 0; i < numElements; ++i){
      if (h_B[i] != h_A[i]){
          fprintf(stderr, "Result verification failed at element %d!\n", i);
          exit(EXIT_FAILURE);
      }
  }
  printf("Test PASSED\n");

  // Free device global memory
  hipFree(d_A);
  hipFree(d_B);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  // Free host memory
  free(h_A);
  free(h_B);

  return 0;
}

